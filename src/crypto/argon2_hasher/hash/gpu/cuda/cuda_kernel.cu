#include "hip/hip_runtime.h"
#include <hip/driver_types.h>

#include <crypto/Argon2_constants.h>

#include "../../../common/common.h"

#include "crypto/argon2_hasher/hash/Hasher.h"
#include "crypto/argon2_hasher/hash/argon2/Argon2.h"

#include "CudaHasher.h"

#define THREADS_PER_LANE                    8
#define BLOCK_SIZE_UINT4                    64
#define BLOCK_SIZE_UINT                     256
#define KERNEL_WORKGROUP_SIZE   		    32
#define ARGON2_PREHASH_DIGEST_LENGTH_UINT   16
#define ARGON2_PREHASH_SEED_LENGTH_UINT     18


#include "blake2b.cu"

#define COMPUTE(alo, ahi, blo, bhi, clo, chi, dlo, dhi)	\
	asm ("{"	\
		".reg .u32 s1, s2, s3, s4;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s1, %0, %6;\n\t"	\
		"xor.b32 %6, %1, %7;\n\t"	\
		"mov.b32 %7, s1;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s4, s3, 24;\n\t"	\
		"shf.r.wrap.b32 %2, s3, s4, 24;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s3, %0, %6;\n\t"	\
		"xor.b32 s4, %1, %7;\n\t"	\
		"shf.r.wrap.b32 %7, s4, s3, 16;\n\t"	\
		"shf.r.wrap.b32 %6, s3, s4, 16;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s3, s4, 31;\n\t"	\
		"shf.r.wrap.b32 %2, s4, s3, 31;\n\t"	\
	"}" : "+r"(alo), "+r"(ahi), "+r"(blo), "+r"(bhi), "+r"(clo), "+r"(chi), "+r"(dlo), "+r"(dhi));

#define G1()           \
{                           \
    COMPUTE(data_a.x, data_a.y, data_c.x, data_c.y, data_e.x, data_e.y, data_g.x, data_g.y) \
    COMPUTE(data_a.z, data_a.w, data_c.z, data_c.w, data_e.z, data_e.w, data_g.z, data_g.w) \
    COMPUTE(data_b.x, data_b.y, data_d.x, data_d.y, data_f.x, data_f.y, data_h.x, data_h.y) \
    COMPUTE(data_b.z, data_b.w, data_d.z, data_d.w, data_f.z, data_f.w, data_h.z, data_h.w) \
}

#define G2()           \
{                           \
    COMPUTE(data_a.x, data_a.y, data_c.z, data_c.w, data_f.x, data_f.y, data_h.z, data_h.w) \
    COMPUTE(data_a.z, data_a.w, data_d.x, data_d.y, data_f.z, data_f.w, data_g.x, data_g.y) \
    COMPUTE(data_b.x, data_b.y, data_d.z, data_d.w, data_e.x, data_e.y, data_g.z, data_g.w) \
    COMPUTE(data_b.z, data_b.w, data_c.x, data_c.y, data_e.z, data_e.w, data_h.x, data_h.y) \
}

#define SHUFFLE() \
{           \
    local_mem[id] = data_a; \
    local_mem[id + 8] = data_b; \
    local_mem[id + 16] = data_c; \
    local_mem[id + 24] = data_d; \
    local_mem[id + 32] = data_e; \
    local_mem[id + 40] = data_f; \
    local_mem[id + 48] = data_g; \
    local_mem[id + 56] = data_h; \
    __syncwarp(); \
    data_a = local_mem[id * 8]; \
    data_b = local_mem[id * 8 + 1]; \
    data_c = local_mem[id * 8 + 2]; \
    data_d = local_mem[id * 8 + 3]; \
    data_e = local_mem[id * 8 + 4]; \
    data_f = local_mem[id * 8 + 5]; \
    data_g = local_mem[id * 8 + 6]; \
    data_h = local_mem[id * 8 + 7]; \
}

inline __host__ __device__ void operator^=( uint4& a, uint4 s) {
   a.x ^= s.x; a.y ^= s.y; a.z ^= s.z; a.w ^= s.w;
}

__global__ void fill_blocks(uint32_t *scratchpad0,
							uint32_t *scratchpad1,
							uint32_t *scratchpad2,
							uint32_t *scratchpad3,
							uint32_t *scratchpad4,
							uint32_t *scratchpad5,
							uint32_t *out,
                            uint32_t *refs, // 32 bit
                            uint32_t *idxs, // first bit is keep flag, next 31 bit is current idx
							uint32_t *segments,
							int memsize,
							int lanes,
                            int seg_length,
                            int seg_count,
							int threads_per_chunk,
							int thread_idx) {
    extern __shared__ uint32_t shared[];
    uint4 data_a, data_b, data_c, data_d, data_e, data_f, data_g, data_h;
    uint4 saved_a, saved_b, saved_c, saved_d, saved_e, saved_f, saved_g, saved_h;

    int session = threadIdx.x / THREADS_PER_LANE;
    int id = threadIdx.x % THREADS_PER_LANE;
    int wave_id = threadIdx.x % 32;

    int local_hash = session % 4;
    int lane = session / 4; // 4 hashes session for each lane
    int base_hash = (blockIdx.x * 4);
    int mem_hash = base_hash + thread_idx;

	int lane_length = seg_length * 4;

    uint4 *local_mem = reinterpret_cast<uint4*>(shared + (lane + local_hash * lanes) * BLOCK_SIZE_UINT);
    uint32_t *local_refs = shared + lanes * 4 * BLOCK_SIZE_UINT + lane * 32;
    uint32_t *local_idxs = shared + lanes * 4 * BLOCK_SIZE_UINT + (lanes + lane) * 32;

    int scratchpad_location = mem_hash / threads_per_chunk;
    uint4 *memory = reinterpret_cast<uint4*>(scratchpad0);
    if(scratchpad_location == 1) memory = reinterpret_cast<uint4*>(scratchpad1);
    if(scratchpad_location == 2) memory = reinterpret_cast<uint4*>(scratchpad2);
    if(scratchpad_location == 3) memory = reinterpret_cast<uint4*>(scratchpad3);
    if(scratchpad_location == 4) memory = reinterpret_cast<uint4*>(scratchpad4);
    if(scratchpad_location == 5) memory = reinterpret_cast<uint4*>(scratchpad5);
    int hash_offset = mem_hash - scratchpad_location * threads_per_chunk;
    memory = memory + hash_offset * (memsize >> 4); // memsize / 16 -> 16 bytes in uint4

	uint4 *next_block;
	uint4 *prev_block;
	uint4 *ref_block;
    uint32_t *seg_refs, *seg_idxs;

    segments += (lane * 3);

	for(int s = 0; s < (seg_count / lanes); s++) {
		int idx = ((s == 0) ? 2 : 0); // index for first slice in each lane is 2
		int with_xor = ((s >= 4) ? 1 : 0);
		int keep = 1;
		int slice = s % 4;
		int pass = s / 4;

		uint32_t *cur_seg = &segments[s * lanes * 3];

		uint32_t cur_idx = cur_seg[0];
        uint32_t prev_idx = cur_seg[1];
        uint32_t seg_type = cur_seg[2];
        uint32_t ref_idx = 0;

        prev_block = memory + prev_idx * BLOCK_SIZE_UINT4 * 4; // 4 hashes are intercalated in a single block

        data_a = prev_block[wave_id];
        data_b = prev_block[wave_id + 32];
        data_c = prev_block[wave_id + 64];
        data_d = prev_block[wave_id + 96];
        data_e = prev_block[wave_id + 128];
        data_f = prev_block[wave_id + 160];
        data_g = prev_block[wave_id + 192];
        data_h = prev_block[wave_id + 224];

        __syncthreads();

        if(seg_type == 0) {
            seg_refs = refs + ((s * lanes + lane) * seg_length - ((s > 0) ? lanes : lane) * 2);
            if(idxs != NULL) seg_idxs = idxs + ((s * lanes + lane) * seg_length - ((s > 0) ? lanes : lane) * 2);

            for (cur_idx--;idx < seg_length; seg_refs += 32, seg_idxs += 32) {
				uint64_t i_limit = seg_length - idx;
				if (i_limit > 32) i_limit = 32;

                local_refs[wave_id] = seg_refs[wave_id];

                if(idxs != NULL) {
                    local_idxs[wave_id] = seg_idxs[wave_id];
                }

                for (int i = 0; i < i_limit; i++, idx++) {
                    ref_idx = local_refs[i];

                    if(idxs != NULL) {
                        cur_idx = local_idxs[i];
                        keep = cur_idx & 0x80000000;
                        cur_idx = cur_idx & 0x7FFFFFFF;
                    }
                    else
                        cur_idx++;

                    ref_block = memory + ref_idx * BLOCK_SIZE_UINT4 * 4;
                    next_block = memory + cur_idx * BLOCK_SIZE_UINT4 * 4;

                    data_a ^= ref_block[wave_id];
                    data_b ^= ref_block[wave_id + 32];
                    data_c ^= ref_block[wave_id + 64];
                    data_d ^= ref_block[wave_id + 96];
                    data_e ^= ref_block[wave_id + 128];
                    data_f ^= ref_block[wave_id + 160];
                    data_g ^= ref_block[wave_id + 192];
                    data_h ^= ref_block[wave_id + 224];

                    saved_a = data_a;
                    saved_b = data_b;
                    saved_c = data_c;
                    saved_d = data_d;
                    saved_e = data_e;
                    saved_f = data_f;
                    saved_g = data_g;
                    saved_h = data_h;

					G1();
                    G2();
                    SHUFFLE();
                    G1();
                    G2();
                    SHUFFLE();

                    if(with_xor == 1) {
                        saved_a ^= next_block[wave_id];
                        saved_b ^= next_block[wave_id + 32];
                        saved_c ^= next_block[wave_id + 64];
                        saved_d ^= next_block[wave_id + 96];
                        saved_e ^= next_block[wave_id + 128];
                        saved_f ^= next_block[wave_id + 160];
                        saved_g ^= next_block[wave_id + 192];
                        saved_h ^= next_block[wave_id + 224];
                    }

                    data_a ^= saved_a;
                    data_b ^= saved_b;
                    data_c ^= saved_c;
                    data_d ^= saved_d;
                    data_e ^= saved_e;
                    data_f ^= saved_f;
                    data_g ^= saved_g;
                    data_h ^= saved_h;

                    if(keep > 0) {
                        next_block[wave_id] = data_a;
                        next_block[wave_id + 32] = data_b;
                        next_block[wave_id + 64] = data_c;
                        next_block[wave_id + 96] = data_d;
                        next_block[wave_id + 128] = data_e;
                        next_block[wave_id + 160] = data_f;
                        next_block[wave_id + 192] = data_g;
                        next_block[wave_id + 224] = data_h;
					}
                }
            }
        }
        else {
            for (; idx < seg_length; idx++, cur_idx++) {
				uint32_t pseudo_rand_lo = __shfl_sync(0xffffffff, data_a.x, local_hash * 8);
				uint32_t pseudo_rand_hi = __shfl_sync(0xffffffff, data_a.y, local_hash * 8);

				uint64_t ref_lane = pseudo_rand_hi % lanes; // thr_cost
				uint32_t reference_area_size = 0;
				if(pass > 0) {
					if (lane == ref_lane) {
						reference_area_size = lane_length - seg_length + idx - 1;
					} else {
						reference_area_size = lane_length - seg_length + ((idx == 0) ? (-1) : 0);
					}
				}
				else {
					if (lane == ref_lane) {
						reference_area_size = slice * seg_length + idx - 1; // seg_length
					} else {
						reference_area_size = slice * seg_length + ((idx == 0) ? (-1) : 0);
					}
				}
				asm("{mul.hi.u32 %0, %1, %1; mul.hi.u32 %0, %0, %2; }": "=r"(pseudo_rand_lo) : "r"(pseudo_rand_lo), "r"(reference_area_size));

				uint32_t relative_position = reference_area_size - 1 - pseudo_rand_lo;

				ref_idx = ref_lane * lane_length + (((pass > 0 && slice < 3) ? ((slice + 1) * seg_length) : 0) + relative_position) % lane_length;

				ref_block = memory + ref_idx * BLOCK_SIZE_UINT4 * 4;
                next_block = memory + cur_idx * BLOCK_SIZE_UINT4 * 4;

                data_a ^= ref_block[wave_id];
                data_b ^= ref_block[wave_id + 32];
                data_c ^= ref_block[wave_id + 64];
                data_d ^= ref_block[wave_id + 96];
                data_e ^= ref_block[wave_id + 128];
                data_f ^= ref_block[wave_id + 160];
                data_g ^= ref_block[wave_id + 192];
                data_h ^= ref_block[wave_id + 224];

                saved_a = data_a;
                saved_b = data_b;
                saved_c = data_c;
                saved_d = data_d;
                saved_e = data_e;
                saved_f = data_f;
                saved_g = data_g;
                saved_h = data_h;

                G1();
                G2();
                SHUFFLE();
                G1();
                G2();
                SHUFFLE();

                if(with_xor == 1) {
                    saved_a ^= next_block[wave_id];
                    saved_b ^= next_block[wave_id + 32];
                    saved_c ^= next_block[wave_id + 64];
                    saved_d ^= next_block[wave_id + 96];
                    saved_e ^= next_block[wave_id + 128];
                    saved_f ^= next_block[wave_id + 160];
                    saved_g ^= next_block[wave_id + 192];
                    saved_h ^= next_block[wave_id + 224];
                }

                data_a ^= saved_a;
                data_b ^= saved_b;
                data_c ^= saved_c;
                data_d ^= saved_d;
                data_e ^= saved_e;
                data_f ^= saved_f;
                data_g ^= saved_g;
                data_h ^= saved_h;

                next_block[wave_id] = data_a;
                next_block[wave_id + 32] = data_b;
                next_block[wave_id + 64] = data_c;
                next_block[wave_id + 96] = data_d;
                next_block[wave_id + 128] = data_e;
                next_block[wave_id + 160] = data_f;
                next_block[wave_id + 192] = data_g;
                next_block[wave_id + 224] = data_h;
            }
        }
	}

    local_mem[id * 8] = data_a;
    local_mem[id * 8 + 1] = data_b;
    local_mem[id * 8 + 2] = data_c;
    local_mem[id * 8 + 3] = data_d;
    local_mem[id * 8 + 4] = data_e;
    local_mem[id * 8 + 5] = data_f;
    local_mem[id * 8 + 6] = data_g;
    local_mem[id * 8 + 7] = data_h;

    __syncthreads();

	// at this point local_mem will contain the final blocks

	if(lane == 0) { // first lane needs to acumulate results
        data_a = make_uint4(0, 0, 0, 0);
        data_b = make_uint4(0, 0, 0, 0);
        data_c = make_uint4(0, 0, 0, 0);
        data_d = make_uint4(0, 0, 0, 0);
        data_e = make_uint4(0, 0, 0, 0);
        data_f = make_uint4(0, 0, 0, 0);
        data_g = make_uint4(0, 0, 0, 0);
        data_h = make_uint4(0, 0, 0, 0);

        local_mem = reinterpret_cast<uint4*>(shared + local_hash * lanes * BLOCK_SIZE_UINT);
		for(int l=0; l<lanes; l++) {
			uint4 *block = local_mem + l * BLOCK_SIZE_UINT4;
            data_a ^= block[id * 8];
            data_b ^= block[id * 8 + 1];
            data_c ^= block[id * 8 + 2];
            data_d ^= block[id * 8 + 3];
            data_e ^= block[id * 8 + 4];
            data_f ^= block[id * 8 + 5];
            data_g ^= block[id * 8 + 6];
            data_h ^= block[id * 8 + 7];
		}

		uint4 *out_mem = reinterpret_cast<uint4*>(out + (base_hash + local_hash) * BLOCK_SIZE_UINT);
        out_mem[id * 8] = data_a;
        out_mem[id * 8 + 1] = data_b;
        out_mem[id * 8 + 2] = data_c;
        out_mem[id * 8 + 3] = data_d;
        out_mem[id * 8 + 4] = data_e;
        out_mem[id * 8 + 5] = data_f;
        out_mem[id * 8 + 6] = data_g;
        out_mem[id * 8 + 7] = data_h;
	}
};

__global__ void prehash (uint32_t *scratchpad0,
                        uint32_t *scratchpad1,
                        uint32_t *scratchpad2,
                        uint32_t *scratchpad3,
                        uint32_t *scratchpad4,
                        uint32_t *scratchpad5,
                        uint32_t *preseed,
                        int memsize,
                        int memcost,
                        int lanes,
                        int passes,
                        int pwdlen,
                        int saltlen,
                        int seg_length,
                        int threads,
                        int threads_per_chunk,
                        int thread_idx) { // len is given in uint32 units
    extern __shared__ uint32_t shared[]; // size = max(lanes * 2, 8) * 88

	int seeds_batch_size = blockDim.x / 4; // number of seeds per block
	int hash_batch_size = seeds_batch_size / (lanes * 2); // number of hashes per block

	int id = threadIdx.x; // minimum 32 threads
	int thr_id = id % 4; // thread id in session
	int session = id / 4; // blake2b hashing session

    int hash_base = blockIdx.x * hash_batch_size;
    int hash_idx = session / (lanes * 2);

    if((hash_base + hash_idx) < threads) {
        int hash_session = session % (lanes * 2); // session in hash

        int lane = hash_session / 2;  // 2 lanes
        int idx = hash_session % 2; // idx in lane

        uint32_t *local_outBuff = &shared[session * BLOCK_SIZE_UINT];
        uint32_t *local_mem = &shared[seeds_batch_size * BLOCK_SIZE_UINT + session * BLAKE_SHARED_MEM_UINT];

        uint64_t *h = (uint64_t *) &local_mem[20];
        uint32_t *buf = (uint32_t *) &h[10];
        uint32_t *value = &buf[32];
        uint32_t *local_preseed = &value[1];

        uint32_t *cursor_in = preseed;
        uint32_t *cursor_out = local_preseed;

        for(int i=0; i < (pwdlen >> 2); i++, cursor_in += 4, cursor_out += 4) {
            cursor_out[thr_id] = cursor_in[thr_id];
        }

        if(thr_id == 0) {
            for (int i = 0; i < (pwdlen % 4); i++) {
                cursor_out[i] = cursor_in[i];
            }

            uint32_t nonce = (preseed[9] >> 24) | (preseed[10] << 8);
            nonce += (hash_base + hash_idx);
            local_preseed[9] = (preseed[9] & 0x00FFFFFF) | (nonce << 24);
            local_preseed[10] = (preseed[10] & 0xFF000000) | (nonce >> 8);
        }

        int buf_len = blake2b_init(h, ARGON2_PREHASH_DIGEST_LENGTH_UINT, thr_id);
        *value = lanes; //lanes
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = 32; //outlen
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = memcost; //m_cost
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = passes; //t_cost
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = ARGON2_VERSION; //version
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = ARGON2_TYPE_VALUE; //type
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        *value = pwdlen * 4; //pw_len
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        buf_len = blake2b_update(local_preseed, pwdlen, h, buf, buf_len, thr_id);
        *value = saltlen * 4; //salt_len
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
		buf_len = blake2b_update(local_preseed, saltlen, h, buf, buf_len, thr_id);
        *value = 0; //secret_len
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);
        *value = 0; //ad_len
        buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
        buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);

        blake2b_final(local_mem, ARGON2_PREHASH_DIGEST_LENGTH_UINT, h, buf, buf_len, thr_id);

        if (thr_id == 0) {
            local_mem[ARGON2_PREHASH_DIGEST_LENGTH_UINT] = idx;
            local_mem[ARGON2_PREHASH_DIGEST_LENGTH_UINT + 1] = lane;
        }

        blake2b_digestLong(local_outBuff, ARGON2_DWORDS_IN_BLOCK, local_mem, ARGON2_PREHASH_SEED_LENGTH_UINT, thr_id,
            &local_mem[20]);

        int mem_hash = hash_base + thread_idx;
        int scratchpad_location = mem_hash / threads_per_chunk;
        uint4 *memory = reinterpret_cast<uint4*>(scratchpad0);
        if(scratchpad_location == 1) memory = reinterpret_cast<uint4*>(scratchpad1);
        if(scratchpad_location == 2) memory = reinterpret_cast<uint4*>(scratchpad2);
        if(scratchpad_location == 3) memory = reinterpret_cast<uint4*>(scratchpad3);
        if(scratchpad_location == 4) memory = reinterpret_cast<uint4*>(scratchpad4);
        if(scratchpad_location == 5) memory = reinterpret_cast<uint4*>(scratchpad5);
        int hash_offset = mem_hash - scratchpad_location * threads_per_chunk;
        memory = memory + hash_offset * (memsize >> 4); // memsize / 16 -> 16 bytes in uint4

        int lane_length = seg_length * 4;

        uint32_t *mem_seed = shared + hash_idx * lanes * 2 * BLOCK_SIZE_UINT;
        uint4 *seed_dst = memory + lane * (lane_length * 4) * BLOCK_SIZE_UINT4; // lane_length * 4 because we intercalate 4 hashes in memory
        uint4 *seed_src = reinterpret_cast<uint4*>(mem_seed + lane * 2 * BLOCK_SIZE_UINT);

        int thr_in_lane = threadIdx.x % THREADS_PER_LANE;

        for(int i=0; i < 8; i++)
        seed_dst[id + i * 32] = seed_src[i + thr_in_lane * 8]; // id * 8 - split the block in 8 succesive regions of 8 uint4 each

        seed_src += BLOCK_SIZE_UINT4;
        seed_dst += (4 * BLOCK_SIZE_UINT4);

        for(int i=0; i < 8; i++)
        seed_dst[id + i * 32] = seed_src[i + thr_in_lane * 8];
    }
}

__global__ void posthash (
        uint32_t *hash,
        uint32_t *out,
        uint32_t *preseed) {
    extern __shared__ uint32_t shared[]; // size = 120

    int hash_id = blockIdx.x;
    int thread = threadIdx.x;

    uint32_t *local_hash = hash + hash_id * ((ARGON2_RAW_LENGTH / 4) + 1);
    uint32_t *local_out = out + hash_id * BLOCK_SIZE_UINT;

    blake2b_digestLong(local_hash, ARGON2_RAW_LENGTH / 4, local_out, ARGON2_DWORDS_IN_BLOCK, thread, shared);

    if(thread == 0) {
        uint32_t nonce = (preseed[9] >> 24) | (preseed[10] << 8);
        nonce += hash_id;
        local_hash[ARGON2_RAW_LENGTH / 4] = nonce;
    }
}

void cuda_allocate(CudaDeviceInfo *device, double chunks, size_t chunk_size) {
	Argon2Profile *profile = device->profileInfo.profile;

	device->error = hipSetDevice(device->cudaIndex);
	if(device->error != hipSuccess) {
		device->errorMessage = "Error setting current device for memory allocation.";
		return;
	}

	size_t allocated_mem_for_current_chunk = 0;

	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_0, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_1, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_2, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_3, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_4, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memoryChunk_5, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}

	uint32_t *refs = (uint32_t *)malloc(profile->blockRefsSize * sizeof(uint32_t));
	for(int i=0;i<profile->blockRefsSize;i++) {
		refs[i] = profile->blockRefs[i*3 + 1];
	}

	device->error = hipMalloc(&device->arguments.refs, profile->blockRefsSize * sizeof(uint32_t));
	if(device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}

	device->error = hipMemcpy(device->arguments.refs, refs, profile->blockRefsSize * sizeof(uint32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->errorMessage = "Error copying memory.";
		return;
	}
	free(refs);

	if(profile->succesiveIdxs == 1) {
		device->arguments.idxs = NULL;
	}
	else {
		uint32_t *idxs = (uint32_t *) malloc(profile->blockRefsSize * sizeof(uint32_t));
		for (int i = 0; i < profile->blockRefsSize; i++) {
			idxs[i] = profile->blockRefs[i * 3];
			if (profile->blockRefs[i * 3 + 2] == 1) {
				idxs[i] |= 0x80000000;
			}
		}

		device->error = hipMalloc(&device->arguments.idxs, profile->blockRefsSize * sizeof(uint32_t));
		if (device->error != hipSuccess) {
			device->errorMessage = "Error allocating memory.";
			return;
		}

		device->error = hipMemcpy(device->arguments.idxs, idxs, profile->blockRefsSize * sizeof(uint32_t),
								   hipMemcpyHostToDevice);
		if (device->error != hipSuccess) {
			device->errorMessage = "Error copying memory.";
			return;
		}
		free(idxs);
	}

	//reorganize segments data
	device->error = hipMalloc(&device->arguments.segments, profile->segCount * 3 * sizeof(uint32_t));
	if(device->error != hipSuccess) {
		device->errorMessage = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments, profile->segments, profile->segCount * 3 * sizeof(uint32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->errorMessage = "Error copying memory.";
		return;
	}

#ifdef PARALLEL_CUDA
	int threads = device->profileInfo.threads / 2;
#else
	int threads = device->profileInfo.threads;
#endif

	size_t preseed_memory_size = profile->pwdLen * 4;
	size_t seed_memory_size = threads * (profile->thrCost * 2) * ARGON2_BLOCK_SIZE;
	size_t out_memory_size = threads * ARGON2_BLOCK_SIZE;
	size_t hash_memory_size = threads * (xmrig::ARGON2_HASHLEN + 4);

    device->error = hipMalloc(&device->arguments.preseedMemory[0], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.seedMemory[0], seed_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.outMemory[0], out_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.hashMemory[0], hash_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.hostSeedMemory[0], 132 * threads);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating pinned memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.preseedMemory[1], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.seedMemory[1], seed_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.outMemory[1], out_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.hashMemory[1], hash_memory_size);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.hostSeedMemory[1], 132 * threads);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error allocating pinned memory.";
        return;
    }
}

void cuda_free(CudaDeviceInfo *device) {
	hipSetDevice(device->cudaIndex);

	if(device->arguments.idxs != NULL) {
		hipFree(device->arguments.idxs);
		device->arguments.idxs = NULL;
	}

	if(device->arguments.refs != NULL) {
		hipFree(device->arguments.refs);
		device->arguments.refs = NULL;
	}

	if(device->arguments.segments != NULL) {
		hipFree(device->arguments.segments);
		device->arguments.segments = NULL;
	}

    if(device->arguments.memoryChunk_0 != NULL) {
        hipFree(device->arguments.memoryChunk_0);
        device->arguments.memoryChunk_0 = NULL;
    }

    if(device->arguments.memoryChunk_1 != NULL) {
        hipFree(device->arguments.memoryChunk_1);
        device->arguments.memoryChunk_1 = NULL;
    }

    if(device->arguments.memoryChunk_2 != NULL) {
        hipFree(device->arguments.memoryChunk_2);
        device->arguments.memoryChunk_2 = NULL;
    }

    if(device->arguments.memoryChunk_3 != NULL) {
        hipFree(device->arguments.memoryChunk_3);
        device->arguments.memoryChunk_3 = NULL;
    }

    if(device->arguments.memoryChunk_4 != NULL) {
        hipFree(device->arguments.memoryChunk_4);
        device->arguments.memoryChunk_4 = NULL;
    }

    if(device->arguments.memoryChunk_5 != NULL) {
        hipFree(device->arguments.memoryChunk_5);
        device->arguments.memoryChunk_5 = NULL;
    }

    if(device->arguments.preseedMemory != NULL) {
        for(int i=0;i<2;i++) {
            if(device->arguments.preseedMemory[i] != NULL)
                hipFree(device->arguments.preseedMemory[i]);
            device->arguments.preseedMemory[i] = NULL;
        }
    }

	if(device->arguments.seedMemory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.seedMemory[i] != NULL)
				hipFree(device->arguments.seedMemory[i]);
			device->arguments.seedMemory[i] = NULL;
		}
	}

	if(device->arguments.outMemory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.outMemory[i] != NULL)
				hipFree(device->arguments.outMemory[i]);
			device->arguments.outMemory[i] = NULL;
		}
	}

    if(device->arguments.hashMemory != NULL) {
        for(int i=0;i<2;i++) {
            if(device->arguments.hashMemory[i] != NULL)
                hipFree(device->arguments.hashMemory[i]);
            device->arguments.hashMemory[i] = NULL;
        }
    }

	if(device->arguments.hostSeedMemory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.hostSeedMemory[i] != NULL)
				hipHostFree(device->arguments.hostSeedMemory[i]);
			device->arguments.hostSeedMemory[i] = NULL;
		}
	}

	hipDeviceReset();
}

bool cuda_kernel_prehasher(void *memory, int threads, Argon2Profile *profile, void *user_data) {
    CudaGpuMgmtThreadData *gpumgmt_thread = (CudaGpuMgmtThreadData *)user_data;
    CudaDeviceInfo *device = gpumgmt_thread->device;
    hipStream_t stream = (hipStream_t)gpumgmt_thread->deviceData;

    int sessions = max(profile->thrCost * 2, (uint32_t)8);
    double hashes_per_block = sessions / (profile->thrCost * 2.0);
    size_t work_items = sessions * 4;

    gpumgmt_thread->lock();

    memcpy(device->arguments.hostSeedMemory[gpumgmt_thread->threadId], memory, gpumgmt_thread->hashData.inSize);

    device->error = hipMemcpyAsync(device->arguments.preseedMemory[gpumgmt_thread->threadId], device->arguments.hostSeedMemory[gpumgmt_thread->threadId], gpumgmt_thread->hashData.inSize, hipMemcpyHostToDevice, stream);
    if (device->error != hipSuccess) {
        device->errorMessage = "Error writing to gpu memory.";
        gpumgmt_thread->unlock();
        return false;
    }

	prehash <<< ceil(threads / hashes_per_block), work_items, sessions * (BLAKE_SHARED_MEM + ARGON2_BLOCK_SIZE), stream>>> (
            (uint32_t*)device->arguments.memoryChunk_0,
            (uint32_t*)device->arguments.memoryChunk_1,
            (uint32_t*)device->arguments.memoryChunk_2,
            (uint32_t*)device->arguments.memoryChunk_3,
            (uint32_t*)device->arguments.memoryChunk_4,
            (uint32_t*)device->arguments.memoryChunk_5,
			device->arguments.preseedMemory[gpumgmt_thread->threadId],
            profile->memSize,
            profile->memCost,
			profile->thrCost,
			profile->segCount / (4 * profile->thrCost),
            gpumgmt_thread->hashData.inSize / 4,
			profile->saltLen,
            profile->segSize,
            threads,
            device->profileInfo.threads_per_chunk,
            gpumgmt_thread->threadsIdx);

    return true;
}

void *cuda_kernel_filler(int threads, Argon2Profile *profile, void *user_data) {
	CudaGpuMgmtThreadData *gpumgmt_thread = (CudaGpuMgmtThreadData *)user_data;
	CudaDeviceInfo *device = gpumgmt_thread->device;
	hipStream_t stream = (hipStream_t)gpumgmt_thread->deviceData;

    size_t work_items = KERNEL_WORKGROUP_SIZE * profile->thrCost;
    size_t shared_mem = profile->thrCost * (4 * ARGON2_BLOCK_SIZE + 128 + (profile->succesiveIdxs == 1 ? 128 : 0));

	fill_blocks <<<threads / 4, work_items, shared_mem, stream>>> ((uint32_t*)device->arguments.memoryChunk_0,
			(uint32_t*)device->arguments.memoryChunk_1,
			(uint32_t*)device->arguments.memoryChunk_2,
			(uint32_t*)device->arguments.memoryChunk_3,
			(uint32_t*)device->arguments.memoryChunk_4,
			(uint32_t*)device->arguments.memoryChunk_5,
			device->arguments.outMemory[gpumgmt_thread->threadId],
			device->arguments.refs,
			device->arguments.idxs,
			device->arguments.segments,
			profile->memSize,
			profile->thrCost,
			profile->segSize,
			profile->segCount,
			device->profileInfo.threads_per_chunk,
            gpumgmt_thread->threadsIdx);

	return (void *)1;
}

bool cuda_kernel_posthasher(void *memory, int threads, Argon2Profile *profile, void *user_data) {
	CudaGpuMgmtThreadData *gpumgmt_thread = (CudaGpuMgmtThreadData *)user_data;
	CudaDeviceInfo *device = gpumgmt_thread->device;
	hipStream_t stream = (hipStream_t)gpumgmt_thread->deviceData;

    size_t work_items = 4;

	posthash <<<threads, work_items, BLAKE_SHARED_MEM, stream>>> (
            device->arguments.hashMemory[gpumgmt_thread->threadId],
            device->arguments.outMemory[gpumgmt_thread->threadId],
            device->arguments.preseedMemory[gpumgmt_thread->threadId]);

	device->error = hipMemcpyAsync(device->arguments.hostSeedMemory[gpumgmt_thread->threadId], device->arguments.hashMemory[gpumgmt_thread->threadId], threads * (xmrig::ARGON2_HASHLEN + 4), hipMemcpyDeviceToHost, stream);
	if (device->error != hipSuccess) {
		device->errorMessage = "Error reading gpu memory.";
		gpumgmt_thread->unlock();
		return false;
	}

	while(hipStreamQuery(stream) != hipSuccess) {
		this_thread::sleep_for(chrono::milliseconds(10));
		continue;
	}

    memcpy(memory, device->arguments.hostSeedMemory[gpumgmt_thread->threadId], threads * (xmrig::ARGON2_HASHLEN + 4));
	gpumgmt_thread->unlock();

	return memory;
}